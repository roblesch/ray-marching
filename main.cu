#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "camera.cuh"
//#include "cloud.cuh"
#include "common.cuh"
//#include "diffuse.cuh"
#include "scene.cuh"
#include "sphere.cuh"
#include "scene.cu"
//#include "cloud.cu"
//#include "diffuse.cu"
//#include "PerlinNoise.cu"

using namespace std;
#define checkCudaErrors(val) CUDA_check_error((val), #val, __FILE__, __LINE__)

void CUDA_check_error(hipError_t res, char const *const func, const char *const file, int const line) {
    if (res) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(res) << " at " << file << ":" << line << " : " << func << "\n";
        hipDeviceReset();
        exit(90);

    }
}

void write_color(std::ostream& out, vec3 pixel_color) {
    // Clamp color to (0.0, 1.0)
    //vclamp(pixel_color);
    //if(pixel_color[0])

    // Write the translated [0,255] value of each color component.
    out << static_cast<int>(255.999 * pixel_color.x()) << ' '
        << static_cast<int>(255.999 * pixel_color.y()) << ' '
        << static_cast<int>(255.999 * pixel_color.z()) << '\n';
}


__global__ void diffuse_scene_full(scene **d_world) {
    // 3 spheres with flat, normals, and diffuse shading
    if(threadIdx.x==0 && blockIdx.x==0){
        (*d_world) = new scene();
        light l = light(
                vec3(1, 1, 1),
                vec3(0.3, 0.3, 0.3));
        light l2 = light(
                vec3(-1, 1, 1),
                vec3(0.4, 0.4, 0.4));
        (**d_world).add_light(l);
        (**d_world).add_light(l2);

        //auto d1 = new diffuse(
        //        vec3(0.3, 0.1, 0.1),
        //        vec3(0.9, 0.2, 0.2),
        //        vec3(0.4, 0.4, 0.4),
        //        vec3(0.2, 0.2, 0.2),
        //        16);

        sphere *s1 = new sphere(
                vec3(-1.1, 0, -2), 0.5, new diffuse());
        sphere *s2 = new sphere(
                vec3(0, 0, -2), 0.5, new diffuse());
        sphere *s3 = new sphere(
                vec3(1.1, 0, -2), 0.5, new diffuse());
        sphere *s4 = new sphere(
                vec3(0, 1.1, -2), 0.5, new diffuse());
        sphere *s5 = new sphere(
                vec3(0, -1.1, -2), 0.5, new diffuse());
        //auto s2 = make_shared<sphere>(
        //       vec3(0, 0, -2), 0.5,
        //        make_shared<normals>());
        //auto s3 = make_shared<perturbed_sphere>(
        //       vec3(1.1, 0, -2), 0.5, 9.0, 0.11, d1);
        //auto box1 = make_shared<box>(
        //        vec3(0, 0, -2), vec3(0.20, 0.20, 0.80), make_shared<normals>());
        //auto csg1 = make_shared<csgObject>(
        //        box1, s2, SUBTRACT, d1);

        //(**d_world).add_surface(*s1);
        (**d_world).add_surface(*s1);
        (**d_world).add_surface(*s2);
        (**d_world).add_surface(*s3);
        (**d_world).add_surface(*s4);
        (**d_world).add_surface(*s5);
        //(**d_world).add_surface(s3);
        //(**d_world).add_surface(csg1);
    }

}

__global__ void diffuse_scene_partial(scene **d_world) {
    // 3 spheres with flat, normals, and diffuse shading
    if(threadIdx.x==0 && blockIdx.x==0){
        (*d_world) = new scene();
        light l = light(
                vec3(1, 1, 1),
                vec3(0.3, 0.3, 0.3));
        light l2 = light(
                vec3(-1, 1, 1),
                vec3(0.4, 0.4, 0.4));
        (**d_world).add_light(l);
        (**d_world).add_light(l2);

        //auto d1 = new diffuse(
        //        vec3(0.3, 0.1, 0.1),
        //        vec3(0.9, 0.2, 0.2),
        //        vec3(0.4, 0.4, 0.4),
        //        vec3(0.2, 0.2, 0.2),
        //        16);

        sphere *s1 = new sphere(
                vec3(0, 0, -2), 0.5, new diffuse());
        //auto s2 = make_shared<sphere>(
        //       vec3(0, 0, -2), 0.5,
        //        make_shared<normals>());
        //auto s3 = make_shared<perturbed_sphere>(
        //       vec3(1.1, 0, -2), 0.5, 9.0, 0.11, d1);
        //auto box1 = make_shared<box>(
        //        vec3(0, 0, -2), vec3(0.20, 0.20, 0.80), make_shared<normals>());
        //auto csg1 = make_shared<csgObject>(
        //        box1, s2, SUBTRACT, d1);

        //(**d_world).add_surface(*s1);
        (**d_world).add_surface(*s1);
        //(**d_world).add_surface(s3);
        //(**d_world).add_surface(csg1);
    }

}

__global__ void render_scene(vec3* fb, int image_x, int image_y,int n_sampling ,camera **d_cam, scene **d_world,hiprandState *d_rand_state) {
    // Render the scene and write to ofs
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if((i>=image_x) || (j>=image_y)) return;
    int pixel_idx = i+ j*image_x;
    hiprandState local_state = d_rand_state[pixel_idx];
    vec3 pixel_color;
    for (int s = 0; s < n_sampling; s++) {
        float u = float(i + hiprand_uniform(&local_state)) / float(image_x);
        float v = float(j + hiprand_uniform(&local_state)) / float(image_y);
        ray r = (**d_cam).get_ray(u, v);
        pixel_color += (**d_world).ray_color(r);
    }
    pixel_color /= n_sampling;
    vclamp(pixel_color);
    fb[pixel_idx] = pixel_color;
    //write_color(ofs, pixel_color);
    //std::cerr << "\nDone.\n";
}

__global__ void camera_init(camera **d_cam){
        if(threadIdx.x==0 && blockIdx.x==0){
                const double fov = 90;
                const double aspect_ratio = 1;
                const vec3 camera_origin(0, 0, 0);
                const vec3 camera_lookat(0, 0, -1);
                const vec3 camera_up(0, 1, 0);
                *d_cam = new camera(camera_origin, camera_lookat, camera_up,fov, aspect_ratio);
        }
}

__global__ void render_init(hiprandState *d_rand_state, int image_x, int image_y){
        int i = ( blockIdx.x*blockDim.x + threadIdx.x), j = ( blockIdx.y*blockDim.y + threadIdx.y);
        if((i>=image_x) || (j>=image_y)) return;
        int pixel_idx = i+ j*image_x;
        hiprand_init(580,pixel_idx,0,&d_rand_state[pixel_idx]);
}

__global__ void free_device_memory(camera **d_cam, scene **d_world){
        //delete((**d_world).lights[0]);
        //delete((**d_world).lights[1]);
        delete((**d_world).surface_list[0]);

}

int main() {

    std::ofstream file;
    file.open("output.ppm");
    int image_x = 256, image_y = 256, thread_x = 128, thread_y = 1, n_sampling = 8;
    //int max_num_obj = 20;
    int pixel_size = image_x * image_y;
    size_t fb_size = pixel_size * sizeof(vec3);


    std::cerr << "Rendering a " << image_x << "x" << image_y << " image with " << n_sampling << " samples per pixel ";
    std::cerr << "in " << thread_x << "x" << thread_y << " blocks.\n";

    

    vec3* FrameBuffer;
    checkCudaErrors(hipMallocManaged((void**)&FrameBuffer, fb_size));

    //cuda random state:
    hiprandState* d_rand_state; 
    checkCudaErrors(hipMalloc((void **)&d_rand_state, pixel_size * sizeof(hiprandState)));


    camera **d_cam;
    scene **d_world;
    checkCudaErrors(hipMalloc((void **)&d_cam, sizeof(camera *)));
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(scene *)));

    camera_init<<<1,1>>>(d_cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    diffuse_scene_partial<<<1,1>>>(d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    clock_t start;
    clock_t end;
    start = clock();
    dim3 threads(thread_x,thread_y);
    dim3 blocks((image_x/thread_x)+1, (image_y/thread_y)+1);
    render_init<<<blocks, threads>>>(d_rand_state, image_x, image_y);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render_scene<<<blocks, threads>>>(FrameBuffer,image_x, image_y, n_sampling,d_cam,d_world,d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    end = clock();
    double time_taken = ((double) (end-start)) / CLOCKS_PER_SEC;
    std::cerr << "Time Taken: " << time_taken << " seconds.\n";

    // Camera
    //camera cam(camera_origin, camera_lookat, camera_up,
    //           fov, aspect_ratio);

    // Scene
    //scene world = diffuse_scene();

    // Render
    //std::ofstream ofs("image.ppm");
    //ofs << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    
    //render_scene(file, cam, world);
    //writing image
    file << "P3\n" << image_y << ' ' << image_x << "\n255\n";
    for (int j = image_y - 1; j >= 0; --j) {
        for (int i = 0; i < image_x; ++i) {
            int pixel_idx = j*image_x + i;
            write_color(file, FrameBuffer[pixel_idx]);
        }

    }

    file.close();
    
    //freeing memory
    free_device_memory<<<1,1>>>(d_cam,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(FrameBuffer));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_cam));
    checkCudaErrors(hipFree(d_rand_state));

    hipDeviceReset();

//#if defined(_WIN32)
//   system("image.ppm");
//#elif defined(__APPLE__)
//    system("open image.ppm");
//#endif

}
