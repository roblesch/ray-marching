#include "hip/hip_runtime.h"
//
// Created by Vansh Dhar on 4/18/22.
//

#include "scene.cuh"
#include "sphere.cuh"

__device__ bool near_zero(double d) {
    // Check if a double is close to 0
    return std::abs(d) < 0.0000000000000001;
}

__device__ double scene::distance_estimator(vec3 p) {
    // Approximate distance between point and nearest surface
    double mindist = INT_MAX;
    nearest = surface_list[0];
    for (int i=0;i<5;i++) {//const surface1& s : surface_list
        sphere* s = surface_list[i];
        double d = s->distance(p);
        if (d < mindist) {
            mindist = d;
            nearest = s;
        }
    }
    return mindist;
}

__device__ bool scene::march(const ray& r, hit_record& rec) {
    // March along a ray by the distance to the nearest surface
    double t = 0;
    while(t < 100) {
        vec3 p = r.at(t);
        double dist = distance_estimator(p);
        if (near_zero(dist) || dist < 0) {
            rec.p = p;
            rec.N = normal(p);
            rec.mat_ptr = nearest->mat_ptr;
            return true;
        }
        dist = dist < 0.001 ? 0.001 : dist;
        t += dist;
    }
    return false;
}

__device__ vec3 scene::normal(const vec3 &p) {
    const vec3 stepx(0.001, 0.0, 0.0);
    const vec3 stepy(0.0, 0.001, 0.0);
    const vec3 stepz(0.0, 0.0, 0.001);

    double gradx = distance_estimator(p + stepx) - distance_estimator(p - stepx);
    double grady = distance_estimator(p + stepy) - distance_estimator(p - stepy);
    double gradz = distance_estimator(p + stepz) - distance_estimator(p - stepz);

    vec3 normal(gradx, grady, gradz);
    return normalize(normal);
}

__device__ vec3 scene::ray_color(const ray& r) {
    // Return the color of a surface at the ray intersection if it hits,
    // otherwise return the color of the background
    hit_record rec;
    if (march(r, rec))
        return rec.mat_ptr->color(r, rec.p, rec.N, lights);
    return background_color(r);
}

__device__ vec3 scene::background_color(const ray &r) {
    auto t = 0.5*(r.direction().y() + 1.0);
    return (1.0-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}
