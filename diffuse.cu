#include "hip/hip_runtime.h"
//
// Created by Vansh Dhar on 4/18/22.
//

#include "diffuse.cuh"

__device__ vec3 flat::color(const ray& r, vec3 p, vec3 N, light lights[20]) const {
    // Flat color
    return albedo;
}

__device__ vec3 normals::color(const ray& r, vec3 p, vec3 N, light lights[20]) const {
    // Color as a function of surface normal
    return 0.5*vec3(N.x()+1, N.y()+1, N.z()+1);
}

__device__ vec3 diffuse::color(const ray& r, vec3 p, vec3 N, light lights[20]) const {
    // Simple diffuse approximation
    vec3 C;

    vec3 E = -r.direction();
    double NE = dot(N, E);

    for ( int i=0;i<2;i++) {//const light &l : lights,
        const light &l = lights[i];
        double NL = dot(N, l.L);
        if (NL * NE < 0) {
            continue;
        }
        if (NL < 0 && NE < 0) {
            N = -N;
            NL = dot(N, l.L);
            NE = dot(N, E);
        }
        vec3 R = 2*NL*N - l.L;
        double RE = dot(R, E);
        RE = clamp(RE);

        C += Ks*l.le*pow(RE,spec) + Kd*l.le*NL;
    }

    C += Ka*la;
    vclamp(C);

    return C;
}
